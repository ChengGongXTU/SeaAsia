#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

#define RADIANCE_RAY_TYPE 0
#define SHADOW_RAY_TYPE 1

#include "../../PathTracerLight.h"
#include "../../random.h"
#include "../../PathTracerMaterial.h"
#include "../../PathReflection.h"
#include "../../PathIntegrator.h"
#include "../../GradientPath.h"

using namespace optix;

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );
rtDeclareVariable(GradientPath, current_prd, rtPayload, );
//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------
rtDeclareVariable(unsigned int, mipmapIndex, , );

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int, loop_depth, , );
rtDeclareVariable(unsigned int, bool_compute_gradients, , );

rtBuffer<float4, 2>					output_buffer;
rtBuffer<DirectionalLight>			dirLights;
rtBuffer<RtMaterial>					materials;
rtBuffer<unsigned int>						texture_id;
rtDeclareVariable(int,         skybox_id, , );

rtDeclareVariable(float3, bg_color, , );

rtTextureSampler<float4, 2> envmap;

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3, texcoord,         attribute texcoord, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );
rtDeclareVariable(unsigned int, primitive_mat_ind, attribute primitive_mat_ind, );
rtTextureSampler<float4, 2> albedoTex;
rtTextureSampler<float4, 2> normalTex;
rtTextureSampler<float4, 2> mraTex;

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void pathtrace_camera()
{	
    size_t2 screen = output_buffer.size();
    float2 inv_screen = 1.0f/make_float2(screen);
	float2 pixel = (make_float2(launch_index) + make_float2(0.5)) * inv_screen;
    float2 jitter_scale = make_float2(1.0f / sqrt_num_samples);
	unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
	
	float3 result = make_float3(0.0);

	//float theta = atan2f(ray.direction.x, ray.direction.z);
	//float phi = M_PIf * 0.5f - acosf(ray.direction.y);
	//float u = (theta + M_PIf) * (0.5f * M_1_PIf);
	//float v = 0.5f * (1.0f + sin(phi));
	//float4 hdrEnv = tex2D(envmap, u, v);
	float u = pixel.x;
	float v = pixel.y;
	float theta = u * 2.0f * M_PIf - M_PIf;
	float phi = asinf(v * 2.0f - 1.0f);
	float y = cosf(M_PIf * 0.5f - phi);
	float xDeivZ = tanf(theta);

	float3 normal;
	//optix::cosine_sample_hemisphere(u, v, normal);
	normal.x = sinf(v * M_PIf) * cosf(u * 2 * M_PIf);
	normal.y = cosf(v * M_PIf);
	normal.z = sinf(v * M_PIf) * sinf(u * 2 * M_PIf);
	normal = normalize(normal);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do 
    {	
		optix::Onb onb(normal);
		float2 uv = make_float2(rnd(seed), rnd(seed));
		float3 direction;
		float rough =  mipmapIndex / 5.0f;
		specular_sample_hemisphere(uv.x, uv.y, direction, rough);
		direction = normalize(direction);
		direction = normalize(LocaltoWorld(direction, onb));
		direction = 2.0f * dot(normal, direction) * direction - normal;

		float theta = atan2f(direction.x, direction.z);
		float phi = M_PIf * 0.5f - acosf(direction.y);
		float u = (theta + M_PIf) * (0.5f * M_1_PIf);
		float v = 0.5f * (1.0f + sin(phi));
		float4 hdrEnv = tex2D(envmap, u, v);
		float3 emitterRadiance = make_float3(hdrEnv.x, hdrEnv.y, hdrEnv.z);
		
		result += fmaxf(dot(direction, normal), 0.0f) * emitterRadiance;
    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

    output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
}

RT_PROGRAM void disneyBRDF()
{

}

  
//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------



RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------


RT_PROGRAM void miss()
{	
	float theta = atan2f( ray.direction.x, ray.direction.z );
	float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
	float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v     = 0.5f * ( 1.0f + sin(phi) );
	float4 hdrEnv = tex2D(envmap, u, v);

    current_prd.emitterRadiance = make_float3(hdrEnv.x, hdrEnv.y, hdrEnv.z);
	current_prd.emitterPdf = 0;
	current_prd.hitType = ENV;
    current_prd.done = true;

	if (current_prd.depth == 0)
	{	
		current_prd.feature.albedo = make_float3(0,0,0);
		current_prd.feature.normal = -ray.direction;
		current_prd.feature.depth = make_float3(1000);
	}
}


