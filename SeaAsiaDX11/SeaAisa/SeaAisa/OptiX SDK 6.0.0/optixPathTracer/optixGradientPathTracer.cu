#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>

#define RADIANCE_RAY_TYPE 0
#define SHADOW_RAY_TYPE 1

#include "../../PathTracerLight.h"
#include "../../random.h"
#include "../../PathTracerMaterial.h"
#include "../../PathReflection.h"
#include "../../PathIntegrator.h"
#include "../../GradientPath.h"

using namespace optix;

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );
rtDeclareVariable(GradientPath, current_prd, rtPayload, );
//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtBuffer<float4, 2>					output_buffer;
rtBuffer<float4, 2>					gradient_dx_buffer;
rtBuffer<float4, 2>					gradient_dy_buffer;
rtBuffer<float4, 2>					gradient_output_buffer;
rtBuffer<float4, 2>					albedo_output_buffer;
rtBuffer<float4, 2>					normal_output_buffer;
rtBuffer<float4, 2>					depth_output_buffer;
rtBuffer<DirectionalLight>			dirLights;
rtBuffer<RtMaterial>					materials;
rtBuffer<unsigned int>						texture_id;
rtDeclareVariable(int,         skybox_id, , );

RT_PROGRAM void pathtrace_camera()
{	
    size_t2 screen = output_buffer.size();
    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 jitter_scale = inv_screen / sqrt_num_samples;
	unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;

	uint2 shiftPixel[4];
	uint2 offsetX = make_uint2(1, 0);
	uint2 offsetY = make_uint2(0, 1);
	uint minIndex = 0;
	//left
	shiftPixel[0] = launch_index -offsetX;
	shiftPixel[0].x = clamp(shiftPixel[0].x, (uint)0, (uint)(screen.x - 1));
	//shiftPixel[0].x = shiftPixel[0].x < minIndex ? minIndex : shiftPixel[0].x;
	//right
	shiftPixel[1] = launch_index + offsetX;
	shiftPixel[1].x = clamp(shiftPixel[1].x, (uint)0, (uint)(screen.x - 1));
	//shiftPixel[1].x = shiftPixel[1].x > (screen.x - 1) ? screen.x - 1 : shiftPixel[1].x;
	//top
	shiftPixel[2] = launch_index + offsetY;
	shiftPixel[2].y = clamp(shiftPixel[2].y, (uint)0, (uint)(screen.y - 1));
	//shiftPixel[2].y = shiftPixel[2].y > (screen.y - 1) ? screen.y - 1 : shiftPixel[2].y;
	//bottom
	shiftPixel[3] = launch_index - offsetY;
	shiftPixel[3].y = clamp(shiftPixel[3].y, (uint)0, (uint)(screen.y - 1));
	//shiftPixel[3].y = shiftPixel[3].y < minIndex ? minIndex : shiftPixel[3].y;
      
	float3 result = make_float3(0.0f);
	Variance albedo;
	Variance normal;
	Variance depth;
	float3 gradients[4];
	float3 shiftResult[4];
	for (int i = 0; i < 4; ++i)
	{	
		gradients[i] = make_float3(0.0f);
		shiftResult[i] = make_float3(0.0f);
	}
	Variance varGradient[4];


    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do 
    {	
        //pixel sample point
        float3 ray_origin = eye;
		float3 ray_direction = RayDifferential(
			samples_per_pixel, sqrt_num_samples, launch_index,
			seed, inv_screen, jitter_scale, U, V, W);

        // Initialze main ray
        GradientPath mainPath;
        InitializeRay(ray_origin, ray_direction, mainPath, seed, true);

		//Initialze shift ray
		GradientPath shiftedPrd[4];
		for (int i = 0; i < 4; ++i) 
		{	
			float3 shift_origin = eye;
			float3 shift_direction =  RayDifferential(
				samples_per_pixel, sqrt_num_samples, shiftPixel[i],
				seed, inv_screen, jitter_scale, U, V, W);

			InitializeRay(shift_origin, shift_direction, shiftedPrd[i], seed + i, false);
			shiftedPrd[i].alphas = (float)sqrt_num_samples / (sqrt_num_samples + sqrt_num_samples);

			//test
			shiftedPrd[i] = mainPath;
			shiftedPrd[i].type = OFFSET_PATH;

		}
		
		bool StopMainPathLoop = false;
		bool StopOffsetPathLoop = false;
		bool isBreak = false;
		int loopCount = 12;
		int doneIndex = loopCount + 1;

		//begin tracing ray
        for(int i = 0; i < loopCount; i++)
        {	
			//store past main path info
			GradientPath lastMainPath = mainPath;

			// main path's intersection from camera
			if (StopMainPathLoop == false)
			{
				Ray ray = make_Ray(
					mainPath.ray.origin, mainPath.ray.direction,
					RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);

				rtTrace(top_object, ray, mainPath);

				//main path : if hit env or emitter or turn off
				if (mainPath.hitType == ENV || mainPath.hitType == EMITTER)
				{
					float3 mainContribute = make_float3(0);
					float weight = 0;

					float bsdfPdf = mainPath.bsdfResult.pdf;
					float previousPathPdf = mainPath.pdf / bsdfPdf;
					float emitterPdf = mainPath.emitterPdf;

					float weightNumerator = previousPathPdf * bsdfPdf;
					float weightDenominator = (previousPathPdf * previousPathPdf)
						* ((emitterPdf * emitterPdf) + (bsdfPdf * bsdfPdf));

					weight = weightNumerator / (scene_epsilon + weightDenominator);
					mainContribute = mainPath.throughput *  mainPath.emitterRadiance;

					AddRadiance(mainPath.radiance, mainContribute, weight);
				}

				//main path : if hit diffsue surface, sample direct light
				if(mainPath.hitType != ENV
					&& mainPath.hitType != HIT_NONE
					&& mainPath.hitType != EMITTER
					&& mainPath.light.lumPdf > 0.f 
					&& fmaxf(mainPath.light.lum) > 0.f)
				{	
					float3 mainLumContri = make_float3(0.0);
					float3 shiftLumContri = make_float3(0.0);
					float weight = 0;

					//main path
					float bsdfPdf = mainPath.light.bsdfResult.pdf;
					float lightPdf = mainPath.light.lumPdf;

					float mainWeightNumerator = mainPath.pdf * lightPdf;
					float mainWeightDenominator = (mainPath.pdf * mainPath.pdf) *
						((lightPdf * lightPdf) + (bsdfPdf * bsdfPdf));
					weight = mainWeightNumerator / (scene_epsilon + mainWeightDenominator);

					mainLumContri =
						mainPath.throughput * mainPath.light.bsdfResult.weight * mainPath.light.lum;

					AddRadiance(mainPath.radiance, mainLumContri, weight);
				}

				if (mainPath.done == true)
				{
					StopMainPathLoop = true;
					doneIndex = i;
				}
			}

			if (StopMainPathLoop == false)
			{	
				//main path :  Russian roulette termination 
				if(mainPath.depth >= rr_begin_depth)
				{
					float pcont = fmaxf(mainPath.throughput / mainPath.pdf);

					if(rnd(mainPath.seed) >= pcont)
						break;
				
					mainPath.throughput /= pcont;
				}
			
				//main path : update para
				mainPath.depth++;
				mainPath.throughput *= mainPath.bsdfResult.weight;
				mainPath.pdf *= mainPath.bsdfResult.pdf;
				mainPath.eta *= mainPath.bsdfResult.eta;
			}

			//main path beyond offset path for one step
			if (i == 0) continue;
			
			//offset path store main path and next main path info
			for (int offInd = 0; offInd < 4; offInd++)
			{
				GradientPath& offsetPath = shiftedPrd[offInd];
				if (offsetPath.isAlive == false) continue;

				offsetPath.mainDirectLight = lastMainPath.light;
				offsetPath.mainIntersect = lastMainPath.intersect;
				offsetPath.mainBsdf = lastMainPath.bsdfResult;

				offsetPath.nextMainIntersect = mainPath.intersect;
				offsetPath.nextMainBsdf = mainPath.bsdfResult;

				offsetPath.nextDone = mainPath.done;
				offsetPath.nextIsAlive = mainPath.isAlive;

				offsetPath.mainHitType = lastMainPath.hitType;
				offsetPath.nextMainHitType = mainPath.hitType;
			}

			// offset path intersect
			for (int offInd = 0; offInd < 4; offInd++)
			{	
				GradientPath& offsetPath = shiftedPrd[offInd];
				if (offsetPath.isAlive == false) continue;
			
				Ray ray = make_Ray(
					offsetPath.ray.origin, offsetPath.ray.direction,
					RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);

				rtTrace(top_object, ray, offsetPath);
			}

			//offset path hit env or faild trace
			for (int offInd = 0; offInd < 4; offInd++)
			{
				GradientPath& offsetPath = shiftedPrd[offInd];
				
				float baseAlpha = offsetPath.alphas;
				float baseAlphaSqr = baseAlpha * baseAlpha;
				float offsetAlphaSqr = (1.f - baseAlpha) * (1.f - baseAlpha);
				float3 mainContribute = make_float3(0);
				float3 offsetContribute = make_float3(0);
				float weight = 0;

				float bsdfPdf = lastMainPath.bsdfResult.pdf;
				float previousPathPdf = lastMainPath.pdf / bsdfPdf;
				float emitterPdf = lastMainPath.emitterPdf;

				float mainWeightNumerator = previousPathPdf * bsdfPdf;
				float mainWeightDenominator = (previousPathPdf * previousPathPdf)
					* ((emitterPdf * emitterPdf) + (bsdfPdf * bsdfPdf));

				//main path has intensity, but offset path don't have
				if (lastMainPath.done == true && (lastMainPath.hitType == ENV || lastMainPath.hitType == EMITTER))
				{

					if (offsetPath.isAlive == false
						&& offsetPath.connectionState != RAY_NOT_CONNECTED)
					{
						weight = mainWeightNumerator * baseAlpha /
							(scene_epsilon + mainWeightDenominator * baseAlphaSqr);

						mainContribute = lastMainPath.throughput * lastMainPath.emitterRadiance;
						offsetContribute = make_float3(0, 0, 0);
					}

					if (offsetPath.isAlive == false
						&& offsetPath.connectionState == RAY_NOT_CONNECTED)
					{
						weight = 1.f /
							(scene_epsilon + lastMainPath.pdf * baseAlphaSqr);
						
						mainContribute = lastMainPath.throughput * lastMainPath.emitterRadiance;
						offsetContribute = make_float3(0, 0, 0);
						offsetPath.connectionState = RAY_CONNECTED;
					}
				}

				if (offsetPath.isAlive == true && (offsetPath.hitType == ENV || offsetPath.hitType == EMITTER)
					&& offsetPath.connectionState != RAY_NOT_CONNECTED
					)
				{	
					float previousOffsetPathPdf = offsetPath.pdf / offsetPath.bsdfResult.pdf;	
					float offsetWeightDenominator = 
						(previousOffsetPathPdf * previousOffsetPathPdf) *
						((offsetPath.emitterPdf * offsetPath.emitterPdf) 
							+ (offsetPath.bsdfResult.pdf * offsetPath.bsdfResult.pdf));				

					weight = mainWeightNumerator * baseAlpha / 
						(scene_epsilon + offsetWeightDenominator * offsetAlphaSqr + mainWeightDenominator * baseAlphaSqr);
					
					mainContribute = lastMainPath.throughput * lastMainPath.emitterRadiance;
					offsetContribute = offsetPath.throughput * offsetPath.emitterRadiance;

					if (offsetPath.hitType == ENV || offsetPath.hitType == EMITTER)	offsetPath.isAlive = false;
				}

				if (offsetPath.isAlive == true && (offsetPath.hitType == ENV || offsetPath.hitType == EMITTER)
					&& offsetPath.connectionState == RAY_NOT_CONNECTED)
				{
					weight = lastMainPath.pdf * baseAlpha /
						(scene_epsilon + offsetPath.pdf * offsetAlphaSqr + lastMainPath.pdf * baseAlphaSqr);

					mainContribute = lastMainPath.throughput * lastMainPath.emitterRadiance;
					offsetContribute = offsetPath.throughput * offsetPath.emitterRadiance;

					if (offsetPath.hitType == ENV || offsetPath.hitType == EMITTER)	offsetPath.isAlive = false;
					offsetPath.connectionState = RAY_CONNECTED;
				}

				AddGradient(offsetPath.gradient, offsetContribute - mainContribute, weight);
				AddRadiance(offsetPath.radiance, offsetContribute, weight);

				//test
				//if (i == 2)	offsetPath.radiance = offsetPath.throughput * offsetPath.emitterRadiance;
			}
			
			////offset path direct lum for main path
			//for (int offInd = 0; offInd < 4; offInd++)
			//{
			//	GradientPath& offsetPath = shiftedPrd[offInd];

			//	float baseAlpha = offsetPath.alphas;
			//	float baseAlphaSqr = baseAlpha * baseAlpha;
			//	float offsetAlphaSqr = (1.f - baseAlpha) * (1.f - baseAlpha);

			//	float3 mainLumContri = make_float3(0.0);
			//	float3 shiftLumContri = make_float3(0.0);
			//	float weight = 0;

			//	float bsdfPdf = lastMainPath.light.bsdfResult.pdf;
			//	float lightPdf = lastMainPath.light.lumPdf;
			//	float mainWeightNumerator = lastMainPath.pdf * lastMainPath.light.lumPdf;
			//	float mainWeightDenominator = (lastMainPath.pdf * lastMainPath.pdf) *
			//		((lightPdf * lightPdf) + (bsdfPdf * bsdfPdf));

			//	if (offsetPath.isAlive == true)
			//	{	
			//		if (offsetPath.hitType != ENV
			//			&& offsetPath.hitType != HIT_NONE
			//			&& offsetPath.hitType != EMITTER)
			//		{
			//			float mainOpposingCosine = 1;
			//			//dot(offsetPath.mainIntersect.normal, offsetPath.mainDirectLight.dir);
			//			float offsetOpposingCosine = 1;
			//			//dot(offsetPath.intersect.normal, offsetPath.light.dir);
			//			float mainDistSquare = offsetPath.mainDirectLight.dist * offsetPath.mainDirectLight.dist;
			//			float offsetDistSquare = offsetPath.light.dist * offsetPath.light.dist;

			//			float jacobian =
			//				fabs((offsetOpposingCosine * mainDistSquare)) /
			//				(scene_epsilon + fabs(mainOpposingCosine * offsetDistSquare));

			//			float offsetWeightDenominator = (jacobian * offsetPath.pdf) * (jacobian * offsetPath.pdf)
			//				* ((offsetPath.light.lumPdf * offsetPath.light.lumPdf)
			//					+ (offsetPath.light.bsdfResult.pdf * offsetPath.light.bsdfResult.pdf));

			//			weight = mainWeightNumerator * baseAlpha /
			//				(scene_epsilon + offsetWeightDenominator * offsetAlphaSqr
			//					+ mainWeightDenominator * baseAlphaSqr);

			//			shiftLumContri = jacobian * offsetPath.throughput
			//				* offsetPath.light.bsdfResult.weight * offsetPath.light.lum;
			//		}

			//	}
			//	else
			//	{
			//		weight = mainWeightNumerator * baseAlpha /
			//			(scene_epsilon + mainWeightDenominator * baseAlphaSqr);
			//	}

			//	AddGradient(offsetPath.gradient, shiftLumContri - mainLumContri, weight);
			//	AddGradient(offsetPath.radiance, shiftLumContri, weight);
			//}
			
			//offset path update para
			for (int offInd = 0; offInd < 4; offInd++)
			{
				GradientPath& offsetPath = shiftedPrd[offInd];
				if (!offsetPath.isAlive) continue;

				//main path :  Russian roulette termination 
				//if (offsetPath.depth >= rr_begin_depth)
				//{
				//	float pcont = fmaxf(offsetPath.throughput / offsetPath.pdf);

				//	if (rnd(offsetPath.seed) >= pcont)
				//	{
				//		TraceFaild(offsetPath, offsetPath.ray.origin,offsetPath.intersect.normal,offsetPath.intersect.uv);
				//		continue;
				//	}
				//	offsetPath.throughput /= pcont;
				//}

				offsetPath.depth++;
				offsetPath.throughput *= offsetPath.bsdfResult.weight *offsetPath.jacobian;
				offsetPath.pdf *= offsetPath.bsdfResult.pdf * offsetPath.jacobian;
				offsetPath.jacobianDet *= offsetPath.jacobian;
			}
	
			if (isBreak 
				|| doneIndex == 0)
				break;

			if (StopMainPathLoop == true)
			{	
				isBreak = true;
				i = i == 0 ? i : i - 1;
			}
        }

        result += mainPath.radiance;
		albedo.addSample(mainPath.feature.albedo);
		normal.addSample(mainPath.feature.normal);
		depth.addSample(mainPath.feature.depth);

		for (int i = 0; i < 4; i++)
		{
			gradients[i] += shiftedPrd[i].gradient;
			shiftResult[i] += shiftedPrd[i].radiance;
			varGradient[i].addSample(shiftedPrd[i].gradient);	
		}

        seed = mainPath.seed;


    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);
	float3 albedo_color = albedo.mean;
	float3 normal_color = normal.mean;
	float3 depth_color = depth.mean;

	normal_color = make_float3(
		CameraTransform(eye, U, V, W).inverse() * make_float4(normal_color, 0));
	normal_color = (normal_color + make_float3(1)) * 0.5;
	depth_color /= 1000.f;

	float3 pixel_gradient_left = gradients[0] / (sqrt_num_samples*sqrt_num_samples);
	float3 pixel_gradient_right = gradients[1] / (sqrt_num_samples*sqrt_num_samples);
	float3 pixel_gradient_top = gradients[2] / (sqrt_num_samples*sqrt_num_samples);
	float3 pixel_gradient_button = gradients[3] / (sqrt_num_samples*sqrt_num_samples);

	float3 color_gradient = shiftResult[0] / (sqrt_num_samples*sqrt_num_samples);

	////tone mapping
	float A = 2.51;
	float B = 0.03;
	float C = 2.43;
	float D = 0.59;
	float E = 0.14;
	float adapted_lumn = 1.0;
	pixel_color *= adapted_lumn;
	pixel_color = (pixel_color * (A * pixel_color + B)) / (pixel_color * (C * pixel_color + D) + E);

	pixel_color = fmaxf(pixel_color, make_float3(0.f));
	pixel_color.x = fmaxf(1.055 * powf(pixel_color.x, 0.416666667) - 0.055f, 0.f);
	pixel_color.y = fmaxf(1.055 * powf(pixel_color.y, 0.416666667) - 0.055f, 0.f);
	pixel_color.z = fmaxf(1.055 * powf(pixel_color.z, 0.416666667) - 0.055f, 0.f);

	color_gradient *= adapted_lumn;
	color_gradient = (color_gradient * (A * color_gradient + B)) / (color_gradient * (C * color_gradient + D) + E);

	color_gradient = fmaxf(color_gradient, make_float3(0.f));
	color_gradient.x = fmaxf(1.055 * powf(color_gradient.x, 0.416666667) - 0.055f, 0.f);
	color_gradient.y = fmaxf(1.055 * powf(color_gradient.y, 0.416666667) - 0.055f, 0.f);
	color_gradient.z = fmaxf(1.055 * powf(color_gradient.z, 0.416666667) - 0.055f, 0.f);


    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {	
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }

	albedo_output_buffer[launch_index] = make_float4(albedo_color, 1.0f);
	normal_output_buffer[launch_index] = make_float4(normal_color, 1.0f);
	depth_output_buffer[launch_index] = make_float4(depth_color, 1.0f);

	gradient_dx_buffer[shiftPixel[0]] += -make_float4(pixel_gradient_left, 1.0f);
	gradient_dx_buffer[launch_index] += make_float4(pixel_gradient_right, 1.0f);
	gradient_dy_buffer[shiftPixel[2]] += -make_float4(pixel_gradient_top, 1.0f);
	gradient_dy_buffer[launch_index] += make_float4(pixel_gradient_button, 1.0f);

	//float3 leftVarGradient = 0.5f * varGradient[0].getSampleVarianceOfMean();
	//float3 rightVarGradient = 0.5f * varGradient[1].getSampleVarianceOfMean();
	//float3 topVarGradient = 0.5f * varGradient[2].getSampleVarianceOfMean();
	//float3 bottomVarGradient = 0.5f * varGradient[3].getSampleVarianceOfMean();

	//gradient_dx_buffer[shiftPixel[0]] += make_float4(leftVarGradient, 1.0f);
	//gradient_dx_buffer[launch_index] += make_float4(rightVarGradient, 1.0f);
	//gradient_dy_buffer[shiftPixel[2]] += make_float4(topVarGradient, 1.0f);
	//gradient_dy_buffer[launch_index] += make_float4(bottomVarGradient, 1.0f);

	gradient_output_buffer[launch_index] = make_float4(color_gradient, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

/*
rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}
*/

//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3, texcoord,         attribute texcoord, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );
rtDeclareVariable(unsigned int, primitive_mat_ind, attribute primitive_mat_ind, );
rtTextureSampler<float4, 2> albedoTex;
rtTextureSampler<float4, 2> normalTex;
rtTextureSampler<float4, 2> mraTex;
  
RT_PROGRAM void disneyBRDF()
{	
	//hit point
    float3 hitpoint = ray.origin + t_hit * ray.direction;

	//bump
	//none

	//material
	RtMaterial mat = materials[primitive_mat_ind];
	
	//texture
	float2 uv = make_float2(texcoord.x, texcoord.y);
	uv = uv * mat.scale + mat.offset;

	float4 albedoBytes = tex2D(albedoTex, uv.x, uv.y);
	float3 albedo = make_float3(
		albedoBytes.x ,
		albedoBytes.y ,
		albedoBytes.z
	);
	float4 normalBytes = tex2D(normalTex, uv.x, uv.y);
	float3 normal = make_float3(
		normalBytes.x ,
		normalBytes.y ,
		normalBytes.z
	);

	normal = 2 * normal - make_float3(1.f);
	normal = normalize(normal);

	//test
    float3 ns = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 ng = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( ns, -ray.direction, ng );
	optix::Onb onb( ffnormal );

	ns = normalize(LocaltoWorld(normal, onb));
	onb = optix::Onb(ns);

	float4 mraBytes = tex2D(mraTex, uv.x, uv.y);
	float3 mra = make_float3(
		mraBytes.x ,
		mraBytes.y ,
		mraBytes.z
	);

		
	//diffuse
	float3 c = mat.colorFactor * albedo;
	float metallicWeight = mat.metallicFactor.x * mra.x;
	float e = mat.eta.x;	//index of refraction
	float strans = mat.strans.x;	// specular transmitted
	float diffuseWeight = (1.f - metallicWeight) * (1.f - strans);
	float dt = mat.dt.x;	// diffuse transmitted
	float rough = mat.roughnessFactor.x * mra.y;
	float lum = luminance(c);
	float3 cTint = lum > 0 ? (c / lum ) : make_float3(1.f, 1.f, 1.f);

	//sheen
	float sheenWeight = mat.sheenWeight.x;
	float sTint = mat.sTint.x;
	float3 cSheen = lerp(make_float3(1 , 1, 1), cTint, sTint);

	//thin and ss Color
	float flat =  mat.flat.x;
	float3 R = diffuseWeight * (1 - flat) * (1 - dt) * c;
	float3 Rss = diffuseWeight * flat * (1 - dt) * c;

	//sheen Color
	float3 Rsheen =  diffuseWeight * sheenWeight * cSheen;

	//microfacet distribution for Trowbridge-Reitz specular
	float anisotropic = mat.anisotropic.x;
	float aspect = sqrtf(1.f - 0.9 * anisotropic);
	float ax = fmaxf(0.001f, rough * rough / aspect);
	float ay = fmaxf(0.001f, rough * rough * aspect);

	//Trowbridge-Reitz specular's constant
	float specTint = mat.specTint.x;
	float3 specMin = lerp(make_float3(1.f), cTint, specTint);
	float3 cSpec0 = lerp(SchlickR0FromEta(e) * specMin, c, metallicWeight);
	//test
	float3 R0 = lerp(specMin * 0.05, c, metallicWeight);

	//Clearcoat
	float cc = mat.cc.x;
	float3 Rcc = make_float3(cc);
	float glossTint = mat.glossTint.x;
	float gloss = lerp(0.1, 0.001, glossTint);
	
	//BTDF
	float3 Rt = strans * 
				make_float3(sqrtf(c.x), sqrtf(c.y), sqrtf(c.z));

	//lambertian transmission
	float3 RLamTrans = dt * c;

	//sample BRDF
	float3 woWorld = normalize(-ray.direction);
	float3 wiWorld = woWorld;
	float pdf = 0;
	float3 f = make_float3(0.f);
	float u0 = rnd(current_prd.seed);
    float u1 = rnd(current_prd.seed);
	float3 wo = normalize(WorldtoLocal(woWorld, onb));
	float3 wi = wo;

	//save for ngpt
	if (current_prd.depth == 0)
	{
		current_prd.feature.albedo = R;
		current_prd.feature.normal = ns;
		current_prd.feature.depth = make_float3(t_hit);;
	}

	//choose a bxdf:
	//0 - DisneyDiffuse(R|D)
	//1 - DisneyRetro(R|D)
	//2 - MicrofacetReflection(R|G)
	//3 - DisneyFakeSS(R|D)
	//4 - DisneySheen(R|D)
	//5 - DisneyClearcoat(R|G)
	//6 - MicrofacetTransmission(T|G)
	//7 - LambertianTransmission(T|D)
	int matchingComps = 6;
	int comp = 0;

	//remap random value
	float2 uRemapped = make_float2(u0, u1);

	// sample illumination from light
	unsigned int nLights = dirLights.size();
	float3 Ld = make_float3(0.0f);	
	if(nLights > 0)
	{	
		float u4 = rnd(current_prd.seed);
		int lightIndex = min((int)(u4 * nLights), nLights - 1);
		current_prd.light.sample = make_float2(rnd(current_prd.seed), rnd(current_prd.seed));
		
		if (current_prd.type == OFFSET_PATH)
		{
			lightIndex = current_prd.mainDirectLight.index;
			current_prd.light.sample = current_prd.mainDirectLight.sample;
		}

		float lightPdf = 1.f / nLights;
		DirectionalLight light = dirLights[lightIndex];
	
		Ld = EstimateDirect(light, woWorld, wo, ng , ns, 
			R, R0, ax, ay, rough, metallicWeight, e,
			Rss,
			Rsheen,
			Rcc, gloss,
			Rt, 1.f, e,
			RLamTrans,
			matchingComps,
			current_prd.light,
			onb );

		current_prd.light.index = lightIndex;
		current_prd.light.lumPdf *= lightPdf;
		current_prd.light.lightIntersect.hitPoint = hitpoint - current_prd.light.dir * current_prd.light.dist;

		//shadow-ray test
		PerRayData_pathtrace_shadow shadow_prd;
		shadow_prd.inShadow = false;
		const float3 L = normalize(-current_prd.light.dir);
		Ray shadow_ray = make_Ray( hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, current_prd.light.dist - scene_epsilon );
		rtTrace(top_object, shadow_ray, shadow_prd);
		if(shadow_prd.inShadow)
		{
			current_prd.light.lum = make_float3(0.0f);
		}
	}

	//sample chosen bxdf
	if(wo.z == 0.f)
	{
		TraceFaild(current_prd, hitpoint, ns, uv);
		return;
	}

	//test: layered-sample multi-brdf
	float totalRatio = 1 + cc;
	float baseRatio = 1 / totalRatio;
	float ccRatio = cc / totalRatio;
	float diffuseRatio = 0.5 *(1.f - metallicWeight);
	float specularRatio = 1 - diffuseRatio;	
	float sheenRatio = sheenWeight /(1 + sheenWeight);
	float ssRatio = flat;
	float lambertRatiao = 0.5;
	float retroRatio = 1 - lambertRatiao;
	
	float pdf4 = baseRatio * diffuseRatio * sheenRatio;
	float pdf3 = baseRatio * diffuseRatio * (1 - sheenRatio) * ssRatio;
	float pdf0 = baseRatio * diffuseRatio * (1 - sheenRatio) * (1 - ssRatio) * lambertRatiao;
	float pdf1 = baseRatio * diffuseRatio * (1 - sheenRatio) * (1 - ssRatio) * retroRatio;
	float pdf2 = baseRatio * specularRatio;
	float pdf5 = ccRatio;

	if(rnd(current_prd.seed) < baseRatio)
	{
		if( rnd(current_prd.seed) < diffuseRatio)
		{		
			if( rnd(current_prd.seed) < sheenRatio)
			{
				comp = 4;
			}
			else
			{
				if( rnd(current_prd.seed) < ssRatio)
				{
					comp = 3;
				}
				else
				{
					if( rnd(current_prd.seed) < lambertRatiao)
					{
						comp = 0;
					}
					else
					{
						comp = 1;
					}
				}
			}
		}
		else
		{
			comp = 2;
		}		
	}
	else
	{
		comp = 5;
	}

	//test
	float cosThetaT;
	float stransRatio = 
		1 - fresnelDielectricExt(CosTheta(wo), cosThetaT, e);//0.5 *strans;
	stransRatio *= strans;
	if(rnd(current_prd.seed) < stransRatio)
	{
		comp = 6;
	}

	//sample_choosen_brdf
	if(comp == 0)	//DisneyDiffuse
	{	
		f = DisneyDiffuseSampleF(wo, wi, R, pdf, uRemapped);
	}
	if(comp == 1)	//DisneyRetro
	{
		f = DisneyRetroSampleF(wo, wi, R, rough, pdf, uRemapped);
	}	
	if(comp == 2)	//gloss reflection
 	{	
		f = DisneyMicrofacetReflectionSampleF(
			wo, wi, R, rough, pdf, uRemapped, ax, ay, R0, metallicWeight, e);
	}
	if(comp == 3)	//DisneyFakeSS
	{
		f = DisneyFakeSSSampleF(wo, wi, Rss, rough, pdf, uRemapped);
	}
	if(comp == 4)	//DisneySheen
	{
		f = DisneySheenSampleF(wo, wi, Rsheen, pdf, uRemapped);
	}
	if(comp == 5)	//DisneyClearcoat
	{
		f = DisneyClearcoatSampleF(wo, wi, Rcc, gloss, pdf, uRemapped);
	}
	if(comp == 6)	//MicrofacetTransmission
	{
		f = DisneyMicrofacetTransmissionSampleF(
			wo, wi, 1.f, e,  
			ax, ay, Rt, 
			pdf, uRemapped);
	}
	if(comp == 7)	//LambertianTransmission
	{
		f = LambertianTransmissionSampleF(
			wo, wi, uRemapped, pdf, RLamTrans);
	}
	if(fmaxf(f) == 0.f)
	{
		TraceFaild(current_prd, hitpoint,ns,uv);
		return;
	}
	wiWorld = normalize(LocaltoWorld(wi, onb));

	//hit type
	if (comp == 7)
	{
		current_prd.hitType = SPECULAR;
	}
	else
	{
		current_prd.hitType = DIFFUSE;
	}

	//offset path reconnection along wi
	if (current_prd.type == OFFSET_PATH)
	{	
		//test
		//if (current_prd.nextMainHitType == HIT_NONE)
		//{
		//	TraceFaild(current_prd);
		//	return;
		//}

		if (current_prd.connectionState == RAY_CONNECTED)
		{
			wiWorld = current_prd.mainBsdf.wi;
			wi = normalize(WorldtoLocal(wiWorld, onb));
			current_prd.jacobian = 1.0;
		}
		else if (current_prd.connectionState == RAY_RECENTLY_CONNECTED)
		{
			wiWorld = current_prd.mainBsdf.wi;
			wi = normalize(WorldtoLocal(wiWorld, onb));
			current_prd.jacobian = 1.0;
			current_prd.connectionState = RAY_CONNECTED;
		}
		else
		{	
			//main path and offset path's current and next hit-points are on the non-specular surface
			//use hit-points contruct the next path
			if (current_prd.hitType != SPECULAR
				&& current_prd.mainHitType != SPECULAR
				&& current_prd.nextMainHitType != SPECULAR)
			{
				//check if next hit is env
				if (current_prd.nextMainHitType == ENV)
				{	
					float3 mainWi = normalize(WorldtoLocal(current_prd.mainBsdf.wi, onb));
					EnvironmentConnection(mainWi, wi, current_prd.jacobian);
					wiWorld = normalize(LocaltoWorld(wi, onb));

					//env visible test
					PerRayData_pathtrace_shadow visibleRay;
					visibleRay.inShadow = false;
					const float3 L = wiWorld;
					Ray visible_ray = make_Ray(hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, 99999.9 - scene_epsilon);
					rtTrace(top_object, visible_ray, visibleRay);
					if (visibleRay.inShadow)
					{
						current_prd.isAlive = false;
						current_prd.done = true;
					}
				}
				//else if (current_prd.nextMainHitType == HIT_NONE // test-meiyong
				//	|| current_prd.mainHitType == HIT_NONE
				//	|| current_prd.hitType == HIT_NONE)
				//{
				//	shiftConnection(current_prd.mainIntersect.hitPoint, current_prd.nextMainIntersect.hitPoint,
				//		current_prd.nextMainIntersect.normal, hitpoint, wiWorld, current_prd.jacobian, scene_epsilon);
				//	wi = normalize(WorldtoLocal(wiWorld, onb));
				//	//env visible test
				//	PerRayData_pathtrace_shadow visibleRay;
				//	visibleRay.inShadow = false;
				//	const float3 L = wiWorld;
				//	Ray visible_ray = make_Ray(hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, 99999.9 - scene_epsilon);
				//	rtTrace(top_object, visible_ray, visibleRay);
				//	if (visibleRay.inShadow)
				//	{
				//		current_prd.isAlive = false;
				//		current_prd.done = true;
				//	}
				//	current_prd.jacobian = 0;
				//}
				//else if (current_prd.hitType == ENV 
				//	|| current_prd.mainHitType == ENV)
				//{
				//	shiftConnection(current_prd.mainIntersect.hitPoint, current_prd.nextMainIntersect.hitPoint,
				//		current_prd.nextMainIntersect.normal, hitpoint, wiWorld, current_prd.jacobian, scene_epsilon);
				//	wi = normalize(WorldtoLocal(wiWorld, onb));
				//	//env visible test
				//	PerRayData_pathtrace_shadow visibleRay;
				//	visibleRay.inShadow = false;
				//	const float3 L = wiWorld;
				//	Ray visible_ray = make_Ray(hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, 99999.9 - scene_epsilon);
				//	rtTrace(top_object, visible_ray, visibleRay);
				//	if (visibleRay.inShadow)
				//	{
				//		current_prd.isAlive = false;
				//		current_prd.done = true;
				//	}
				//	current_prd.jacobian = 0;
				//}
				else
				{
					shiftConnection(current_prd.mainIntersect.hitPoint, current_prd.nextMainIntersect.hitPoint,
						current_prd.nextMainIntersect.normal, hitpoint, wiWorld, current_prd.jacobian,scene_epsilon);
					wi = normalize(WorldtoLocal(wiWorld, onb));

					//env visible test
					PerRayData_pathtrace_shadow visibleRay;
					visibleRay.inShadow = false;
					const float3 L = wiWorld;
					float dist = length(hitpoint - current_prd.nextMainIntersect.hitPoint);
					Ray visible_ray = make_Ray(hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, dist - scene_epsilon);
					rtTrace(top_object, visible_ray, visibleRay);
					if (visibleRay.inShadow)
					{
						current_prd.isAlive = false;
						current_prd.done = true;
					}

					//test
					//float3 mainDir = current_prd.mainIntersect.hitPoint - current_prd.nextMainIntersect.hitPoint;
					//float mainDistSqr = length(mainDir);
					//mainDistSqr *= mainDistSqr;
					//mainDir = normalize(mainDir);
					//float mainCosine = fabs(dot(mainDir, current_prd.nextMainIntersect.normal));

					//float3 shiftDir = hitpoint - current_prd.nextMainIntersect.hitPoint;
					//float shiftDistSqr = length(shiftDir);
					//shiftDistSqr *= shiftDistSqr;
					//shiftDir = normalize(shiftDir);
					//float shiftCosine = fabs(dot(shiftDir, current_prd.nextMainIntersect.normal));

					//if ((mainDistSqr) == 0.f)
					//{
					//	current_prd.jacobian = 1;
					//}
				}
				current_prd.connectionState = RAY_RECENTLY_CONNECTED;
			}
			// main path and offset path hit specular/transmittance surface
			//use current hitpoint's half-vector  contruct the next path
			else
			{	
				float3 mainWo = normalize(WorldtoLocal(current_prd.mainBsdf.wo, onb));
				float3 mainWi = normalize(WorldtoLocal(current_prd.mainBsdf.wi, onb));
				
				float shiftEta = e / 1.f;
				HalfVectorReconnect(mainWo, mainWi, wo, wi,
					current_prd.mainBsdf.eta, shiftEta, current_prd.jacobian, scene_epsilon);
				wiWorld = normalize(LocaltoWorld(wi, onb));
			}
		}
	}

	//Compute overall PDF with all matching _BxDF_spec

	//test
	float transPdf = 0;
	float3 transF = make_float3(0);
	float3 wi2;
	transF = DisneyMicrofacetTransmissionSampleF(
			wo, wi2, 1.f, e,  
			ax, ay, Rt, 
			transPdf, uRemapped);

	pdf = 0.0;
	pdf += DisneyRetroPdf(wo, wi) * pdf0;
	pdf += DisneyDiffusePdf(wo, wi)* pdf1;
	pdf += DisneyMicrofacetReflectionPdf(wo, wi, ax, ay)* pdf2;
	pdf += DisneyFakeSSPdf(wo, wi)* pdf3;
	pdf += DisneySheenPdf(wo, wi)* pdf4;
	pdf += DisneyClearcoatPdf(wo, wi, gloss)* pdf5;

	//test
	pdf = pdf * (1 - stransRatio);
	pdf += transPdf * stransRatio;

	// Compute value of BSDF for sampled direction
	f = make_float3(0.0);
	f = DisneyBsdfF(woWorld, wiWorld, 
			wo, wi, 
			ns , R, 
			R0, ax, ay, 
			rough, metallicWeight, e,
			Rss,
			Rsheen,
			Rcc, gloss,
			Rt, 1.f, e,
			RLamTrans);

	//test
	f += transF;
	
	if(pdf == 0.f || (f.x == 0.f && f.y == 0.f && f.z == 0.f))
	{	
		TraceFaild(current_prd, hitpoint, ns, uv);
	}
	else
	{
		current_prd.ray.origin = hitpoint;
		current_prd.ray.direction = wiWorld;
		current_prd.countEmitted = false;

		current_prd.intersect.hitPoint = hitpoint;
		current_prd.intersect.normal = ns;
		current_prd.intersect.uv = uv;

		current_prd.bsdfResult.wo = woWorld;
		current_prd.bsdfResult.wi = wiWorld;
		current_prd.bsdfResult.compIndex = comp;
		current_prd.bsdfResult.f = f;
		current_prd.bsdfResult.eta = e / 1.f;
		current_prd.bsdfResult.pdf = pdf;
		current_prd.bsdfResult.weight = f * abs(dot(wiWorld, ns));

		current_prd.emitterRadiance = make_float3(0);
		current_prd.emitterPdf = 0;
	}
}

//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void miss()
{	
	float theta = atan2f( ray.direction.x, ray.direction.z );
	float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
	float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v     = 0.5f * ( 1.0f + sin(phi) );
	float4 hdrEnv = tex2D(envmap, u, v);

    current_prd.emitterRadiance = make_float3(hdrEnv.x, hdrEnv.y, hdrEnv.z);
	current_prd.emitterPdf = 0;
	current_prd.hitType = ENV;
    current_prd.done = true;

	if (current_prd.depth == 0)
	{	
		current_prd.feature.albedo = make_float3(0,0,0);
		current_prd.feature.normal = -ray.direction;
		current_prd.feature.depth = make_float3(1000);
	}
}


