#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2018, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>

#define RADIANCE_RAY_TYPE 0
#define SHADOW_RAY_TYPE 1

#include "../../PathTracerLight.h"
#include "../../random.h"
#include "../../PathTracerMaterial.h"
#include "../../PathReflection.h"
#include "../../PathIntegrator.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  sqrt_num_samples, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );

rtBuffer<float4, 2>					output_buffer;
rtBuffer<DirectionalLight>			dirLights;
rtBuffer<RtMaterial>					materials;
rtBuffer<unsigned int>						texture_id;
rtDeclareVariable(int,         skybox_id, , );

RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float2 jitter_scale = inv_screen / sqrt_num_samples;
    unsigned int samples_per_pixel = sqrt_num_samples*sqrt_num_samples;
    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
    do 
    {
        //
        // Sample pixel using jittering
        //
        unsigned int x = samples_per_pixel%sqrt_num_samples;
        unsigned int y = samples_per_pixel/sqrt_num_samples;
        float2 jitter = make_float2(x-rnd(seed), y-rnd(seed));
        float2 d = pixel + jitter*jitter_scale;
        float3 ray_origin = eye;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;
		float3 total_attenuation = make_float3(1, 1, 1);

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(int i = 0; i < 100; i++)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, RADIANCE_RAY_TYPE, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            if(prd.done)
            {
                // We have hit the background or a luminaire
                prd.result += prd.radiance * total_attenuation;
                break;
            }

			
            // Russian roulette termination 
            if(prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(total_attenuation );
                if(rnd(prd.seed) >= pcont)
                    break;
               total_attenuation /= pcont;
            }
			

            prd.depth++;
            prd.result += prd.radiance * total_attenuation;

			total_attenuation *= prd.attenuation;

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;
        seed = prd.seed;
    } while (--samples_per_pixel);

    //
    // Update the output buffer
    //
    float3 pixel_color = result/(sqrt_num_samples*sqrt_num_samples);

	//tonemapping
	//tone mapping
	float A = 2.51;
	float B = 0.03;
	float C = 2.43;
	float D = 0.59;
	float E = 0.14;
	float adapted_lumn = 1.0;
	pixel_color *= adapted_lumn;
	pixel_color = (pixel_color * (A * pixel_color + B)) / (pixel_color * (C * pixel_color + D) + E);

	pixel_color = fmaxf(pixel_color, make_float3(0.f));
	pixel_color.x = fmaxf(1.055 * powf(pixel_color.x, 0.416666667) - 0.055f, 0.f);
	pixel_color.y = fmaxf(1.055 * powf(pixel_color.y, 0.416666667) - 0.055f, 0.f);
	pixel_color.z = fmaxf(1.055 * powf(pixel_color.z, 0.416666667) - 0.055f, 0.f);


    if (frame_number > 1)
    {
        float a = 1.0f / (float)frame_number;
        float3 old_color = make_float3(output_buffer[launch_index]);
        output_buffer[launch_index] = make_float4( lerp( old_color, pixel_color, a ), 1.0f );
    }
    else
    {	
        output_buffer[launch_index] = make_float4(pixel_color, 1.0f);
    }
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

/*
rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}
*/

//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,     diffuse_color, , );
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable( float3, texcoord,         attribute texcoord, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );
rtDeclareVariable(unsigned int, primitive_mat_ind, attribute primitive_mat_ind, );
rtTextureSampler<float4, 2> albedoTex;
rtTextureSampler<float4, 2> normalTex;
rtTextureSampler<float4, 2> mraTex;
  
RT_PROGRAM void disneyBRDF()
{	

	//normal
    //float3 ns = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
   // float3 ng = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    //float3 ffnormal = faceforward( ns, -ray.direction, ng );
	//optix::Onb onb( ffnormal );

	//hit point
    float3 hitpoint = ray.origin + t_hit * ray.direction;

	//bump
	//none

	//material
	RtMaterial mat = materials[primitive_mat_ind];
	
	//texture
	float2 uv = make_float2(texcoord.x, texcoord.y);
	uv = uv * mat.scale + mat.offset;

	float4 albedoBytes = tex2D(albedoTex, uv.x, uv.y);
	float3 albedo = make_float3(
		albedoBytes.x ,
		albedoBytes.y ,
		albedoBytes.z
	);
	float4 normalBytes = tex2D(normalTex, uv.x, uv.y);
	float3 normal = make_float3(
		normalBytes.x ,
		normalBytes.y ,
		normalBytes.z
	);

	normal = 2 * normal - make_float3(1.f);
	normal = normalize(normal);
	//normal = make_float3(normal.x,normal.z, normal.y);

	//test
    float3 ns = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 ng = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( ns, -ray.direction, ng );
	optix::Onb onb( ffnormal );

	ns = normalize(LocaltoWorld(normal, onb));
	onb = optix::Onb(ns);

	//test
	//current_prd.attenuation = make_float3(1.0);
	//current_prd.radiance =  ns;
	//current_prd.countEmitted = false;
	//current_prd.done = true;
	//return;

	float4 mraBytes = tex2D(mraTex, uv.x, uv.y);
	float3 mra = make_float3(
		mraBytes.x ,
		mraBytes.y ,
		mraBytes.z
	);

		
	//diffuse
	float3 c = mat.colorFactor * albedo;
	float metallicWeight = mat.metallicFactor.x * mra.x;
	float e = mat.eta.x;	//index of refraction
	float strans = mat.strans.x;	// specular transmitted
	float diffuseWeight = (1.f - metallicWeight) * (1.f - strans);
	float dt = mat.dt.x;	// diffuse transmitted
	float rough = mat.roughnessFactor.x * mra.y;
	float lum = luminance(c);
	float3 cTint = lum > 0 ? (c / lum ) : make_float3(1.f, 1.f, 1.f);

	//sheen
	float sheenWeight = mat.sheenWeight.x;
	float sTint = mat.sTint.x;
	float3 cSheen = lerp(make_float3(1 , 1, 1), cTint, sTint);

	//thin and ss Color
	float flat =  mat.flat.x;
	float3 R = diffuseWeight * (1 - flat) * (1 - dt) * c;
	float3 Rss = diffuseWeight * flat * (1 - dt) * c;

	//sheen Color
	float3 Rsheen =  diffuseWeight * sheenWeight * cSheen;

	//microfacet distribution for Trowbridge-Reitz specular
	float anisotropic = mat.anisotropic.x;
	float aspect = sqrtf(1.f - 0.9 * anisotropic);
	float ax = fmaxf(0.001f, rough * rough / aspect);
	float ay = fmaxf(0.001f, rough * rough * aspect);

	//Trowbridge-Reitz specular's constant
	float specTint = mat.specTint.x;
	float3 specMin = lerp(make_float3(1.f), cTint, specTint);
	float3 cSpec0 = lerp(SchlickR0FromEta(e) * specMin, c, metallicWeight);
	//test
	float3 R0 = lerp(specMin * 0.05, c, metallicWeight);

	//Clearcoat
	float cc = mat.cc.x;
	float3 Rcc = make_float3(cc);
	float glossTint = mat.glossTint.x;
	float gloss = lerp(0.1, 0.001, glossTint);
	
	//BTDF
	float3 Rt = strans * 
				make_float3(sqrtf(c.x), sqrtf(c.y), sqrtf(c.z));

	//lambertian transmission
	float3 RLamTrans = dt * c;

	//sample BRDF
	float3 woWorld = normalize(-ray.direction);
	float3 wiWorld = woWorld;
	float pdf = 0;
	float3 f = make_float3(0.f);
	float u0 = rnd(current_prd.seed);
    float u1 = rnd(current_prd.seed);
	float3 wo = normalize(WorldtoLocal(woWorld, onb));
	float3 wi = wo;

	//choose a bxdf:
	//0 - DisneyDiffuse(R|D)
	//1 - DisneyRetro(R|D)
	//2 - MicrofacetReflection(R|G)
	//3 - DisneyFakeSS(R|D)
	//4 - DisneySheen(R|D)
	//5 - DisneyClearcoat(R|G)
	//6 - MicrofacetTransmission(T|G)
	//7 - LambertianTransmission(T|D)
	int matchingComps = 6;

	//test//float u3 = rnd(current_prd.seed);
	//test//int comp = floor(matchingComps * u3);
	int comp = 0;

	//remap random value
	float2 uRemapped = make_float2(u0, u1);

	// sample illumination from light
    unsigned int nLights = dirLights.size();
    float3 Ld = make_float3(0.0f);	
    if(nLights > 0)
	{
		float u4 =rnd(current_prd.seed);
		int lightIndex = min((int)(u4 * nLights), nLights - 1);
		float lightPdf = 1.f / nLights;
		DirectionalLight light = dirLights[lightIndex];
		float3 lightDir;
		float lightDist;

		Ld = EstimateDirect(light, woWorld, wo, ng , ns, 
			R, R0, ax, ay, rough, metallicWeight, e,
			Rss,
			Rsheen,
			Rcc, gloss,
			Rt, 1.f, e,
			RLamTrans,
			matchingComps,lightDir, lightDist, onb ) / lightPdf;

		//shadow-ray test
		PerRayData_pathtrace_shadow shadow_prd;
		shadow_prd.inShadow = false;
		const float3 L = normalize(-lightDir);
		Ray shadow_ray = make_Ray( hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, lightDist - scene_epsilon );
		rtTrace(top_object, shadow_ray, shadow_prd);
		if(shadow_prd.inShadow)
		{
			Ld = make_float3(0.0f);	
		}
	}
	current_prd.radiance = Ld;

	//sample chosen bxdf
	if(wo.z == 0.f)
	{
		current_prd.attenuation = make_float3(1.0);
		current_prd.radiance = make_float3(0.f);
		current_prd.countEmitted = false;
		current_prd.done = true;
		return;
	}

	//test: layered-sample multi-brdf
	float totalRatio = 1 + cc;
	float baseRatio = 1 / totalRatio;
	float ccRatio = cc / totalRatio;
	float diffuseRatio = 0.5 *(1.f - metallicWeight);
	float specularRatio = 1 - diffuseRatio;	
	float sheenRatio = sheenWeight /(1 + sheenWeight);
	float ssRatio = flat;
	float lambertRatiao = 0.5;
	float retroRatio = 1 - lambertRatiao;
	
	float pdf4 = baseRatio * diffuseRatio * sheenRatio;
	float pdf3 = baseRatio * diffuseRatio * (1 - sheenRatio) * ssRatio;
	float pdf0 = baseRatio * diffuseRatio * (1 - sheenRatio) * (1 - ssRatio) * lambertRatiao;
	float pdf1 = baseRatio * diffuseRatio * (1 - sheenRatio) * (1 - ssRatio) * retroRatio;
	float pdf2 = baseRatio * specularRatio;
	float pdf5 = ccRatio;

	if(rnd(current_prd.seed) < baseRatio)
	{
		if( rnd(current_prd.seed) < diffuseRatio)
		{		
			if( rnd(current_prd.seed) < sheenRatio)
			{
				comp = 4;
			}
			else
			{
				if( rnd(current_prd.seed) < ssRatio)
				{
					comp = 3;
				}
				else
				{
					if( rnd(current_prd.seed) < lambertRatiao)
					{
						comp = 0;
					}
					else
					{
						comp = 1;
					}
				}
			}
		}
		else
		{
			comp = 2;
		}		
	}
	else
	{
		comp = 5;
	}

	//test
	float cosThetaT;
	float stransRatio = 
		1 - fresnelDielectricExt(CosTheta(wo), cosThetaT, e);//0.5 *strans;
	stransRatio *= strans;
	if(rnd(current_prd.seed) < stransRatio)
	{
		comp = 6;
	}

	//sample_choosen_brdf
	if(comp == 0)	//DisneyDiffuse
	{	
		f = DisneyDiffuseSampleF(wo, wi, R, pdf, uRemapped);
	}
	if(comp == 1)	//DisneyRetro
	{
		f = DisneyRetroSampleF(wo, wi, R, rough, pdf, uRemapped);
	}	
	if(comp == 2)	//gloss reflection
 	{	
		f = DisneyMicrofacetReflectionSampleF(
			wo, wi, R, rough, pdf, uRemapped, ax, ay, R0, metallicWeight, e);
	}
	if(comp == 3)	//DisneyFakeSS
	{
		f = DisneyFakeSSSampleF(wo, wi, Rss, rough, pdf, uRemapped);
	}
	if(comp == 4)	//DisneySheen
	{
		f = DisneySheenSampleF(wo, wi, Rsheen, pdf, uRemapped);
	}
	if(comp == 5)	//DisneyClearcoat
	{
		f = DisneyClearcoatSampleF(wo, wi, Rcc, gloss, pdf, uRemapped);
	}
	if(comp == 6)	//MicrofacetTransmission
	{
		f = DisneyMicrofacetTransmissionSampleF(
			wo, wi, 1.f, e,  
			ax, ay, Rt, 
			pdf, uRemapped);
	}
	if(comp == 7)	//LambertianTransmission
	{
		f = LambertianTransmissionSampleF(
			wo, wi, uRemapped, pdf, RLamTrans);
	}
	if(fmaxf(f) == 0.f)
	{
		current_prd.attenuation = make_float3(1.0);
		current_prd.done = true;
		return;
	}
	wiWorld = normalize(LocaltoWorld(wi, onb));

	//Compute overall PDF with all matching _BxDF_spec

	//test
	float transPdf = 0;
	float3 transF = make_float3(0);
	float3 wi2;
	transF = DisneyMicrofacetTransmissionSampleF(
			wo, wi2, 1.f, e,  
			ax, ay, Rt, 
			transPdf, uRemapped);

	pdf = 0.0;
	pdf += DisneyRetroPdf(wo, wi) * pdf0;
	pdf += DisneyDiffusePdf(wo, wi)* pdf1;
	pdf += DisneyMicrofacetReflectionPdf(wo, wi, ax, ay)* pdf2;
	pdf += DisneyFakeSSPdf(wo, wi)* pdf3;
	pdf += DisneySheenPdf(wo, wi)* pdf4;
	pdf += DisneyClearcoatPdf(wo, wi, gloss)* pdf5;

	//test
	pdf = pdf * (1 - stransRatio);
	pdf += transPdf * stransRatio;

	//pdf += LambertianTransmissionPdf(wo, wi);
	//test//pdf /= matchingComps;
	
	// Compute value of BSDF for sampled direction
	f = make_float3(0.0);
	f = DisneyBsdfF(woWorld, wiWorld, 
			wo, wi, 
			ns , R, 
			R0, ax, ay, 
			rough, metallicWeight, e,
			Rss,
			Rsheen,
			Rcc, gloss,
			Rt, 1.f, e,
			RLamTrans);

	//test
	f += transF;
	
	if(pdf > 0.f && (f.x > 0.f || f.y > 0.f || f.z > 0.f))
	{	
		current_prd.attenuation *= f * abs(dot(wiWorld, ns)) / pdf;
		current_prd.origin = hitpoint;
		current_prd.direction = wiWorld;
		current_prd.countEmitted = false;
	}
	else
	{
		current_prd.attenuation = make_float3(1.0);
		current_prd.radiance = make_float3(0.f);
		current_prd.countEmitted = false;
		current_prd.done = true;
		return;
	}
}

RT_PROGRAM void DisneyGGXBRDF()
{	
	//normal
    float3 ns = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 ng = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( ns, -ray.direction, ng );
	optix::Onb onb( ffnormal );

	//hit point
    float3 hitpoint = ray.origin + t_hit * ray.direction;

	current_prd.origin = hitpoint;

	//material
	RtMaterial mat = materials[primitive_mat_ind];
	
	//texture
	float4 albedoBytes = tex2D(albedoTex, texcoord.x, texcoord.y);
	float3 albedo = make_float3(
		albedoBytes.x ,
		albedoBytes.y ,
		albedoBytes.z
	);
	float4 normalBytes = tex2D(normalTex, texcoord.x, texcoord.y);
	float3 normal = make_float3(
		normalBytes.x ,
		normalBytes.y ,
		normalBytes.z
	);

	normal = 2 * normal - make_float3(1.f);
	ns = normalize(LocaltoWorld(normal, onb));
	onb = optix::Onb(ns);

	float4 mraBytes = tex2D(mraTex, texcoord.x, texcoord.y);
	float3 mra = make_float3(
		mraBytes.x ,
		mraBytes.y ,
		mraBytes.z
	);

	//diffuse
	float3 c = mat.colorFactor * albedo;
	float metallicWeight = mat.metallicFactor.x * mra.x;
	float e = 1.5f;	//index of refraction
	float strans = 0.f;	// specular transmitted
	float diffuseWeight = (1.f - metallicWeight) * (1.f - strans);
	float dt = 0.f;	// diffuse transmitted
	float rough = mat.roughnessFactor.x * mra.y;
	float lum = luminance(c);
	//normalize lum
	float3 cTint = lum > 0 ? (c / lum ) : make_float3(1.f, 1.f, 1.f);
	float specTint = 0.0f;
	float3 specMin = lerp(make_float3(1.f), cTint, specTint);
	float3 cSpec0 = lerp(SchlickR0FromEta(e) * specMin, c, metallicWeight);

	//test
	cSpec0 = lerp(make_float3(0.05), c, metallicWeight);

	float3 R0 = cSpec0;
	float3 R = diffuseWeight * c;

	//sample para
	float3 woWorld = normalize(-ray.direction);
	float3 wiWorld = woWorld;
	float pdf = 0;
	float3 f = make_float3(0.f);
	float u0 =rnd(current_prd.seed);
    float u1 =rnd(current_prd.seed);
	float2 randomU = make_float2(u0, u1);
	float3 wo = normalize(WorldtoLocal(woWorld, onb));
	float3 wi = wo;

	// sample illumination from light
    unsigned int nLights = dirLights.size();
    float3 Ld = make_float3(0.0f);	
    if(nLights > 0)
	{
		float u3 =rnd(current_prd.seed);
		int lightIndex = min((int)(u3 * nLights), nLights - 1);
		float lightChoosePdf = 1.f / nLights;
		DirectionalLight light = dirLights[lightIndex];

		float3 wiWorldL;
		float3 lightDir;
		float lightDist;
		float lightPdf = 0.f, scatteringPdf = 0;

		float3 Li = LightSampleLi(wiWorldL, lightPdf, light, lightDist);
		float3 wiL = normalize(WorldtoLocal(wiWorldL, onb));
		lightDir = -wiWorldL;

		if(lightPdf > 0)
		{
			float3 fL = DisneyBaseDiffuseF(wo, wiL, R, rough);
			fL += SmithGGXF(wo, wiL, rough, R0);

			Ld += fL * Li *  abs(dot(wiWorldL, ns))/ (lightChoosePdf * lightPdf);
		}

		//shadow-ray test
		PerRayData_pathtrace_shadow shadow_prd;
		shadow_prd.inShadow = false;
		const float3 L = normalize(-lightDir);
		Ray shadow_ray = make_Ray( hitpoint, L, SHADOW_RAY_TYPE, scene_epsilon, lightDist - scene_epsilon );
		rtTrace(top_object, shadow_ray, shadow_prd);
		if(shadow_prd.inShadow)
		{
			Ld = make_float3(0.0f);	
		}
	}
	current_prd.radiance = Ld;

	//sample bxdf for wi direction
	float difProbability = rnd(current_prd.seed);
	float diffuseRatio = 0.5 *(1.f - metallicWeight);

	if (difProbability < diffuseRatio)
	{
		f = DisneyDiffuseSampleF(wo, wi, R, pdf, randomU);
	}
	else
	{
		f = SmithGGXSampleF(wo,wi,R,rough,pdf,randomU,R0);
		if(fmaxf(f) == 0.f)
		{
			current_prd.attenuation = make_float3(1.0);
			current_prd.done = true;
			return;
		}
	}
	wiWorld = normalize(LocaltoWorld(wi, onb));
	current_prd.direction = wiWorld;
	current_prd.origin = current_prd.origin + current_prd.direction * scene_epsilon;
	current_prd.countEmitted = false;

	//pdf
	float specularRatio = 1.f - diffuseRatio;
	float pdfDiff = DisneyDiffusePdf(wo, wi) + DisneyRetroPdf(wo, wi);
	pdfDiff *= 0.5;
	float pdfSpec = SmithGGXPdf(wo, wi, rough);
	pdf = diffuseRatio * pdfDiff + specularRatio * pdfSpec;

	//f
	float3 fDiff = DisneyBaseDiffuseF(wo, wi, R, rough);
	float3 fSpec = SmithGGXF(wo, wi, rough, R0);
	f = fDiff + fSpec;


	if(pdf > 0.0)
	{
		current_prd.attenuation = f * abs(dot(wiWorld, ns)) / pdf;
	}
	else
	{
		current_prd.attenuation = make_float3(1.0);
		current_prd.radiance = make_float3(0.f);
		current_prd.done = true;
		return;
	}

}

//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

rtTextureSampler<float4, 2> envmap;
RT_PROGRAM void miss()
{	
	float theta = atan2f( ray.direction.x, ray.direction.z );
	float phi   = M_PIf * 0.5f -  acosf( ray.direction.y );
	float u     = (theta + M_PIf) * (0.5f * M_1_PIf);
	float v     = 0.5f * ( 1.0f + sin(phi) );
	float4 hdrEnv = tex2D(envmap, u, v);
    current_prd.radiance = make_float3(hdrEnv.x, hdrEnv.y, hdrEnv.z);
    current_prd.done = true;
}


